#include "hip/hip_runtime.h"
// subtractive_attention/subtractive_attention_kernel.cu

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel for subtractive_attention
__global__ void subtractive_attention_kernel(
    const float* __restrict__ inputs,
    const float* __restrict__ keys,
    float* __restrict__ similarities,
    int batch_size,
    int seq_len,
    int num_tokens,
    int channels
) {
    int b = blockIdx.x;
    int s = blockIdx.y;
    int t = blockIdx.z * blockDim.x + threadIdx.x;

    if (b < batch_size && s < seq_len && t < num_tokens) {
        float sum = 0.0f;
        for (int c = 0; c < channels; ++c) {
            float diff = inputs[b * seq_len * channels + s * channels + c] - keys[t * channels + c];
            sum += 1.0f - fabsf(diff);
        }
        similarities[b * seq_len * num_tokens + s * num_tokens + t] = sum;
    }
}

// Launcher function
void subtractive_attention_cuda(
    torch::Tensor inputs,
    torch::Tensor keys,
    torch::Tensor similarities
) {
    const int batch_size = inputs.size(0);
    const int seq_len = inputs.size(1);
    const int num_tokens = keys.size(0);
    const int channels = inputs.size(2);

    // Maximum threads per block (1024 for modern GPUs)
    const int max_threads = 1024;

    // Calculate threads and blocks for token dimension
    const int threads = min(num_tokens, max_threads);
    const int token_blocks = (num_tokens + threads - 1) / threads;

    dim3 blocks(batch_size, seq_len, token_blocks);

    subtractive_attention_kernel<<<blocks, threads>>>(
        inputs.data_ptr<float>(),
        keys.data_ptr<float>(),
        similarities.data_ptr<float>(),
        batch_size,
        seq_len,
        num_tokens,
        channels
    );

    // Check for any CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in subtractive_attention_kernel: %s\n", hipGetErrorString(err));
    }
}
